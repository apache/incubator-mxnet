/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */
#include <mshadow/tensor.h>
#include "./index_array-inl.h"

namespace mxnet {
namespace op {

using namespace mshadow::cuda;

void IndexArrayForwardGPU(const nnvm::NodeAttrs &attrs,
                          const OpContext &ctx,
                          const std::vector<TBlob> &inputs,
                          const std::vector<OpReqType> &req,
                          const std::vector<TBlob> &outputs) {
  using namespace mshadow;
  CHECK_EQ(inputs.size(), 1U);
  CHECK_EQ(outputs.size(), 1U);
  CHECK_EQ(req.size(), 1U);
  const TBlob& in_data = inputs[0];
  const TBlob& out_data = outputs[0];

  const IndexArrayParam& param = nnvm::get<IndexArrayParam>(attrs.parsed);

  const TShape inshape = in_data.shape_;
  const int ndim = inshape.ndim();

  Stream<gpu> *s = ctx.get_stream<gpu>();
  hipStream_t stream = Stream<gpu>::GetStream(s);

  using namespace mxnet_op;

  if (param.axes.has_value()) {
    const mxnet::Tuple<int>& axes = param.axes.value();
    const int naxes = axes.ndim();

    MXNET_IDX_TYPE_SWITCH(param.dtype, DType, {
      std::vector<DType> index_products = IndexArrayComputeIndexProducts<DType>(inshape);

      std::vector<DType> cpu_workspace(2 * naxes);
      IndexArrayBuildSelectedAxesWorkspace(axes, index_products, cpu_workspace.data(), ndim);

      Tensor<gpu, 1, DType> workspace =
          ctx.requested[0].get_space_typed<gpu, 1, DType>(Shape1(2 * naxes), s);

      CUDA_CALL(hipMemcpyAsync(workspace.dptr_, cpu_workspace.data(), sizeof(DType) * (2 * naxes),
                           hipMemcpyHostToDevice, stream));

      // Assumes param.target_axis is -1 or 0.
      const ptrdiff_t index_axis_offset = param.target_axis == -1 ? naxes : 1;
      const ptrdiff_t target_axis_offset = param.target_axis == -1 ? 1: in_data.Size();

      MXNET_ASSIGN_REQ_SWITCH(req[0], req_type, {
        Kernel<IndexArrayKernel<req_type>, gpu>::Launch(s, in_data.Size(),
            out_data.dptr<DType>(), naxes, index_axis_offset, target_axis_offset, workspace.dptr_);
      });
    });
  } else {
    Tensor<gpu, 1, dim_t> workspace =
        ctx.requested[0].get_space_typed<gpu, 1, dim_t>(Shape1(ndim), s);

    CUDA_CALL(hipMemcpyAsync(workspace.dptr_, inshape.data(), sizeof(dim_t) * ndim,
        hipMemcpyHostToDevice, stream));

    // Assumes param.target_axis is -1 or 0.
    const ptrdiff_t index_axis_offset = param.target_axis == -1 ? ndim : 1;
    const ptrdiff_t target_axis_offset = param.target_axis == -1 ? 1: in_data.Size();

    MXNET_IDX_TYPE_SWITCH(param.dtype, DType, {
      MXNET_ASSIGN_REQ_SWITCH(req[0], req_type, {
        Kernel<IndexArrayDefaultKernel<req_type>, gpu>::Launch(s, in_data.Size(),
                                                               out_data.dptr<DType>(), ndim,
                                                               index_axis_offset,
                                                               target_axis_offset,
                                                               workspace.dptr_);
      });
    });
  }
}

NNVM_REGISTER_OP(_contrib_index_array)
.set_attr<FCompute>("FCompute<gpu>", IndexArrayForwardGPU);

}  // namespace op
}  // namespace mxnet
