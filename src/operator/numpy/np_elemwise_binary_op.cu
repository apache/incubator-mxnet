#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2019 by Contributors
 * \file np_elemwise_binary_op.cu
 * \brief GPU Implementation for element-wise binary operators.
 */


#include <mxnet/base.h>
#include "../mshadow_op.h"
#include "../operator_common.h"
#include "../tensor/elemwise_binary_op.h"
#include "../tensor/elemwise_binary_broadcast_op.h"
#include "../tensor/elemwise_binary_scalar_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(ldexp)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastCompute<gpu, mshadow_op::ldexp>);

NNVM_REGISTER_OP(ldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::ldexp>);

NNVM_REGISTER_OP(rldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Compute<gpu, mshadow_op::rldexp>);

NNVM_REGISTER_OP(_backward_ldexp)
.set_attr<FCompute>("FCompute<gpu>", BinaryBroadcastBackwardUseIn<gpu, mshadow_op::ldexp_grad,
                                                              mshadow_op::ldexp_rgrad>);

NNVM_REGISTER_OP(_backward_ldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<gpu, mshadow_op::ldexp_grad>);

NNVM_REGISTER_OP(_backward_rldexp_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarOp::Backward<gpu, mshadow_op::rldexp_grad>);

}  // namespace op
}  // namespace mxnet