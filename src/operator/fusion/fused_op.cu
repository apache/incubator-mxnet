#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

#include <sys/stat.h>
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <nnvm/pass_functions.h>
#include <algorithm>
#include <mutex>
#include "./fused_op.h"
#include "./fused_op-inl.h"
#include "../operator_common.h"
#include "../elemwise_op_common.h"
#include "../../executor/exec_pass.h"
#include "../../common/cuda_utils.h"

namespace mxnet {

namespace detail {

std::string FindCUDAIncludePath() {
#if defined(_WIN32)
  const std::string delimiter = "\\";
#else
  const std::string delimiter = "/";
#endif
  std::string cuda_include_path;
  const char* cuda_path_env = std::getenv("CUDA_PATH");
  if (cuda_path_env != nullptr) {
    cuda_include_path += cuda_path_env;
    cuda_include_path += delimiter + "include";
    return cuda_include_path;
  }

#if defined(__linux__)
  struct stat st;
  cuda_include_path = "/usr/local/cuda/include";
  if (stat(cuda_include_path.c_str(), &st) == 0) {
    return cuda_include_path;
  }
#endif
  LOG(FATAL) << "Cannot find cuda include path."
             << "CUDA_PATH is not set or CUDA is not installed in the default installation path."
             << "In other than linux, it is necessary to set CUDA_PATH.";
  return cuda_include_path;
}


inline std::string mshadowTypeToString(int type) {
  switch (type) {
    case mshadow::kFloat32:
      return "float";
    case mshadow::kFloat64:
      return "double";
    case mshadow::kFloat16:
      return "half";
    case mshadow::kUint8:
      return "unsigned char";
    case mshadow::kInt8:
      return "char";
    case mshadow::kInt32:
      return "int";
    case mshadow::kInt64:
      return "long long";
    default:
      LOG(FATAL) << "Unknown type enum " << type;
  }
  return "";
}

inline int mshadowTypeToVectorLength(int type) {
  switch (type) {
    case mshadow::kFloat32:
      return 1;
    case mshadow::kFloat64:
      return 1;
    case mshadow::kFloat16:
      return 2;
    case mshadow::kUint8:
      return 4;
    case mshadow::kInt8:
      return 4;
    case mshadow::kInt32:
      return 1;
    case mshadow::kInt64:
      return 1;
    default:
      LOG(FATAL) << "Unknown type enum " << type;
  }
  return 0;
}


}  // namespace detail

void FusedOp::GenerateCode(const std::vector<OpReqType> &req) {
  const auto& g = this->symbol_.indexed_graph();
  std::string code = "";
  int temp_name_counter = 0;
  using NodeEntry = nnvm::IndexedGraph::NodeEntry;
  std::map<std::pair<int, int>, std::string> variables;
  std::map<int, int> load_index;

  std::vector<uint32_t> outputs(g.num_nodes());

  for (size_t i = 0; i < g.num_nodes(); ++i) {
    const auto& node = g[i];
    if (node.source != nullptr) {
      outputs[i] = node.source->num_outputs();
    } else {
      outputs[i] = 0;
    }
  }

  for (size_t i = 0; i < g.num_nodes(); ++i) {
    const auto& node = g[i];
    const auto* source = node.source;
    if (source != nullptr) {
        if (source->is_variable()) {
            load_index[i] = 1;
        } else {
            std::string op_name = source->op()->name;
            if (detail::fused_op_slice_ops.find(op_name) != detail::fused_op_slice_ops.end()) {
                load_index[node.inputs[0].node_id] = 0;
            }
        }
    }
  }
  for (size_t i = 0; i < g.num_nodes(); ++i) {
    const auto& node = g[i];
    const auto* source = node.source;
    if (source != nullptr) {
        if (source->is_variable()) {
            if (load_index[i]) {
              const auto& var_name = source->attrs.name;
              code += "const auto vec_" + var_name + " = load_index<nvec>(" + \
                       var_name + ", offset);\n";
              variables[{i, 0}] = var_name;
            }
            CHECK_EQ(outputs[i], 1);
        } else {
            std::string op_name = source->op()->name;
            if (detail::fused_op_slice_ops.find(op_name) != detail::fused_op_slice_ops.end()) {
                int arg_id = node.inputs[0].node_id;
                const auto& var_name = g[arg_id].source->attrs.name;
                load_index[arg_id] = 0;
                std::string begin = source->attrs.dict.at("begin");
                std::string end = source->attrs.dict.at("end");
                if (end == "None") {
                    end = "((1<<31)-1)";
                }
                std::string axis = source->attrs.dict.at("axis");
                const auto vec_name = "vec_" + var_name + "_" + std::to_string(i);
                code += "const auto " + vec_name + " = load_slice<nvec, "+ axis + ">(" + \
                        var_name + ", " + var_name + "_strides," + begin + \
                        "," + end + ", offset);\n";
                CHECK_EQ(outputs[i], 1);
                variables[{i, 0}] = vec_name;
                continue;
            }
        }
    }
  }

  int counter = 0;
  for (const auto& entry : g.outputs()) {
    const auto var_name = "output" + std::to_string(counter);
    code += "VectorType<remove_pointer<decltype(" + var_name + \
            ")>::type, nvec> vec_output" + std::to_string(counter) + ";\n";
    ++counter;
  }

  code += "for (int j = 0; j < nvec; j++ ) {\n";


  for (size_t i = 0; i < g.num_nodes(); ++i) {
    const auto& node = g[i];
    const auto* source = node.source;
    if (source != nullptr) {
      std::string var_name = "temp" + std::to_string(temp_name_counter++);
      if (source->is_variable()) {
        if (load_index[i]) {
            code += "const auto " + var_name + " = load(vec_" + variables[{i, 0}] + ".x[j]);\n";
            CHECK_EQ(outputs[i], 1);
            variables[{i, 0}] = var_name;
        }
      } else {
        std::string op_name = source->op()->name;
        if (detail::fused_op_binary_ops.find(op_name) != detail::fused_op_binary_ops.end()) {
          std::string op = detail::fused_op_binary_ops.at(op_name);
          const auto& arg1 = variables[{node.inputs[0].node_id, node.inputs[0].index}];
          const auto& arg2 = variables[{node.inputs[1].node_id, node.inputs[1].index}];
          code += "const auto " + var_name + " = " + op +
                  "(" + arg1 + ", " + arg2 + ");\n";
          CHECK_EQ(outputs[i], 1);
          variables[{i, 0}] = var_name;
          continue;
        }

        if (detail::fused_op_unary_ops.find(op_name) != detail::fused_op_unary_ops.end()) {
          std::string op = detail::fused_op_unary_ops.at(op_name);
          const auto& arg1 = variables[{node.inputs[0].node_id, node.inputs[0].index}];
          code += "const auto " + var_name + " = " + op +
                  "(" + arg1 + ");\n";
          CHECK_EQ(outputs[i], 1);
          variables[{i, 0}] = var_name;
          continue;
        }

        if (detail::fused_op_special_ops.find(op_name) != detail::fused_op_special_ops.end()) {
          const std::vector<std::string>& op_desc = detail::fused_op_special_ops.at(op_name);
          std::string fmt = op_desc[0];
          for (size_t j = 1; j < op_desc.size(); ++j) {
            const std::string& desc = op_desc[j];
            std::string sub;
            if (desc[0] == '_') {
              // Argument
              int arg_id = std::stoi(desc.substr(1));
              sub = variables[{node.inputs[arg_id].node_id, node.inputs[arg_id].index}];
            } else {
              sub = source->attrs.dict.at(desc);
            }
            size_t pos = fmt.find("%");
            CHECK_NE(pos, std::string::npos);
            fmt.replace(pos, 1, sub);
          }
          code += "const auto " + var_name + " = " + fmt + ";\n";
          CHECK_EQ(outputs[i], 1);
          variables[{i, 0}] = var_name;
          continue;
        }

        if (detail::fused_op_mimo_ops.find(op_name) != detail::fused_op_mimo_ops.end()) {
          const std::vector<std::vector<std::string>>& op_descs =
            detail::fused_op_mimo_ops.at(op_name);
          CHECK_EQ(outputs[i], op_descs.size());
          size_t count = 0;
          for (const auto& op_desc : op_descs) {
            var_name = "temp" + std::to_string(temp_name_counter++);
            std::string fmt = op_desc[0];
            for (size_t j = 1; j < op_desc.size(); ++j) {
              const std::string& desc = op_desc[j];
              std::string sub;
              if (desc[0] == '_') {
                // Argument
                int arg_id = std::stoi(desc.substr(1));
                sub = variables[{node.inputs[arg_id].node_id, node.inputs[arg_id].index}];
              } else {
                sub = source->attrs.dict.at(desc);
              }
              size_t pos = fmt.find("%");
              CHECK_NE(pos, std::string::npos);
              fmt.replace(pos, 1, sub);
            }
            code += "const auto " + var_name + " = " + fmt + ";\n";
            variables[{i, count}] = var_name;
            ++count;
          }
          continue;
        }

        if (detail::fused_op_slice_ops.find(op_name) != detail::fused_op_slice_ops.end()) {
          code += "const auto " + var_name + " = load(" + variables[{i, 0}] + ".x[j]);\n";
          variables[{i, 0}] = var_name;
          continue;
        }


        // Special cases with variable number
        // of inputs/outputs, listed in
        // detail::fused_op_variable_io_ops
        if (op_name == "add_n") {
          CHECK_EQ(outputs[i], 1);
          const auto& arg = variables[{node.inputs[0].node_id, node.inputs[0].index}];
          code += "auto " + var_name + " = " + arg + ";\n";
          for (size_t inp = 1; inp < node.inputs.size(); ++inp) {
            const auto& temp_arg = variables[{node.inputs[inp].node_id, node.inputs[inp].index}];
            code += var_name + " = add(" + var_name + ", " + temp_arg + ");\n";
          }
          variables[{i, 0}] = var_name;
          continue;
        }

        if (op_name == "_backward_Activation") {
          CHECK_EQ(outputs[i], 1);
          std::string act_type = node.source->attrs.dict.at("act_type");
          std::string rhs, lhs;
          rhs = variables[{node.inputs[0].node_id, node.inputs[0].index}];
          if (act_type == "relu" ||
              act_type == "sigmoid" ||
              act_type == "tanh") {
            lhs = variables[{node.inputs[1].node_id, node.inputs[1].index}];
          } else {
            lhs = variables[{node.inputs[2].node_id, node.inputs[2].index}];
          }
          code += "const auto " + var_name + " = backward_" + act_type +
                  "(" + lhs + ", " + rhs + ");\n";

          variables[{i, 0}] = var_name;
          continue;
        }
        LOG(FATAL) << "Unrecognized op " + op_name;
      }
    } else {
      LOG(FATAL) << "Encountered node with NULL source.";
    }
  }

  counter = 0;
  for (const auto& entry : g.outputs()) {
    const std::string& var = variables[{entry.node_id, entry.index}];
    const auto var_name = "output" + std::to_string(counter);
    code += "vec_" + var_name + ".x[j] = store("+ var +", " + var_name + ");\n";
    ++counter;
  }

  code += "}\n";

  counter = 0;

  for (const auto& entry : g.outputs()) {
    const std::string& var = variables[{entry.node_id, entry.index}];
    if (req[counter] == kWriteTo || req[counter] == kWriteInplace) {
      const auto var_name = "output" + std::to_string(counter);
      code += "store_index(vec_" + var_name + ", i, " + var_name + ");\n";
    } else if (req[counter] == kAddTo) {
      const auto var_name = "output" + std::to_string(counter);
      code += "store_add_index(vec_" + var_name + ", i, " + var_name + ");\n";
    } else if (req[counter] == kNullOp) {
      // NULL req, do not do anything
    } else {
      LOG(FATAL) << "Encountered unexpected req.";
    }
    ++counter;
  }

  this->code_ = code;
}

template <>
void FusedOp::Forward<gpu>(const nnvm::NodeAttrs& attrs,
                           const OpContext &ctx,
                           const std::vector<TBlob> &inputs,
                           const std::vector<OpReqType> &req,
                           const std::vector<TBlob> &outputs) {
  using namespace mshadow;
  std::lock_guard<std::mutex> lock(my_mutex_);
  CHECK_GE(outputs.size(), 1) << "There needs to be at least 1 output.";

  std::vector<int> in_dtypes;
  std::vector<int> in_ndims;
  std::vector<int> out_dtypes;
  int ndim = outputs[0].ndim();
  int nvec = 1;

  size_t counter = 0;
  for (const auto& blob : inputs) {
    in_dtypes.push_back(blob.type_flag_);
    in_ndims.push_back(blob.ndim());
    initialized_ = initialized_ && (blob.type_flag_ == inputs_[counter].dtype);
    inputs_[counter].dtype = blob.type_flag_;
    nvec = max(nvec, detail::mshadowTypeToVectorLength(blob.type_flag_));
    ++counter;
  }

  counter = 0;
  for (const auto& blob : outputs) {
    out_dtypes.push_back(blob.type_flag_);
    initialized_ = initialized_ && (blob.type_flag_ == outputs_[counter].dtype);
    outputs_[counter].dtype = blob.type_flag_;
    nvec = max(nvec, detail::mshadowTypeToVectorLength(blob.type_flag_));
    ++counter;
  }

  // Get compute capability of the current GPU
  int dev_id = ctx.run_ctx.ctx.dev_id;
  int cc_major = ComputeCapabilityMajor(dev_id);
  int cc_minor = ComputeCapabilityMinor(dev_id);

  initialized_ = initialized_ && cc_major == this->cc_major_;
  initialized_ = initialized_ && cc_minor == this->cc_minor_;
  this->cc_major_ = cc_major;
  this->cc_minor_ = cc_minor;

  initialized_ = initialized_ && (req == saved_reqs_);
  saved_reqs_ = req;

  if (!initialized_) {
    this->GenerateCode(req);
    LOG(INFO) << code_;
    std::string aux_code = "";
    std::string kernel_params = "";
    std::string tensor_params = "";
    nnvm::Symbol sym;
    sym.outputs = this->symbol_.outputs;
    const std::vector<std::string> input_names = sym.ListInputNames(nnvm::Symbol::kAll);
    size_t num_params = in_dtypes.size() + out_dtypes.size();
    size_t i = 0;
    aux_code += "static const int nvec = " + std::to_string(nvec) + ";\n";
    for (const auto &type : in_dtypes) {
      std::string type_name = detail::mshadowTypeToString(type);
      std::string dtype_var = "DType" + std::to_string(i);
      std::string dim_var = "ndim" + std::to_string(i);
      aux_code = "using " + dtype_var + " = " + type_name + ";\n" + aux_code;
      aux_code = "static const int " + dim_var + " = " + \
                  std::to_string(in_ndims[i]) + ";\n" + aux_code;
      tensor_params += dtype_var + "* " +input_names[i];
      kernel_params += " const Strides<" + dim_var + "> " + input_names[i]+"_strides";
      ++i;
      if (i < num_params) {
        tensor_params += ", ";
        kernel_params += ", ";
      }
    }
    for (const auto &type : out_dtypes) {
      std::string type_name = detail::mshadowTypeToString(type);
      std::string dtype_var = "DType" + std::to_string(i);
      aux_code = "using " + dtype_var + " = " + type_name + ";\n" + aux_code;
      tensor_params += dtype_var + "* output" +
                       std::to_string(i - in_dtypes.size());
      ++i;
      if (i < num_params) {
        tensor_params += ", ";
      }
    }
    kernel_params += tensor_params;
    code_ = std::string(detail::fp16_support_string) + "\n" +
            detail::type_support_string + "\n" +
            detail::fused_op_function_definitions + "\n" +
            aux_code + "\n" +
            "__global__ void FusedKernel_" + attrs.name +
            "(size_t N, " + kernel_params + ") {\n" +
            detail::fused_op_kernel_begin + "\n" +
            code_ + "\n" +
            detail::fused_op_kernel_end;
    // Guard NVRTC calls
    std::lock_guard<std::mutex> lock_nvrtc(mutex_);
    hiprtcProgram program;
    NVRTC_CALL(
        hiprtcCreateProgram(&program,                                 // prog
                           &code_[0],                                // buffer
                           (attrs.name + "_kernel.cu").c_str(),      // name
                           0,                                        // numHeaders
                           NULL,                                     // headers
                           NULL));                                   // includeNames
    std::string gpu_arch = "--gpu-architecture=compute_" +
                           std::to_string(this->cc_major_) +
                           std::to_string(this->cc_minor_);
    std::string cuda_include_path = "-I" + detail::FindCUDAIncludePath();

    const char *opts[] = {gpu_arch.c_str(),
                          "--std=c++11",
                          "-default-device",
                          cuda_include_path.c_str()};
    const std::string kernel_name_demangled = "FusedKernel_" + attrs.name;
    NVRTC_CALL(hiprtcAddNameExpression(program, (kernel_name_demangled).c_str()));

    hiprtcResult compileResult = hiprtcCompileProgram(program,  // prog
                                                    4,        // numOptions
                                                    opts);    // options
    // Obtain compilation log from the program.
    size_t logSize;
    NVRTC_CALL(hiprtcGetProgramLogSize(program, &logSize));
    std::string log(logSize, '\0');
    NVRTC_CALL(hiprtcGetProgramLog(program, &log[0]));
    CHECK_EQ(compileResult, HIPRTC_SUCCESS) << "NVRTC Compilation failed.\n" << log;
    // Obtain PTX from the program.
    size_t ptxSize;
    NVRTC_CALL(hiprtcGetCodeSize(program, &ptxSize));
    ptx_.reserve(ptxSize);
    NVRTC_CALL(hiprtcGetCode(program, &ptx_[0]));
    const char *name;
    NVRTC_CALL(hiprtcGetLoweredName(program,
                                   kernel_name_demangled.c_str(),
                                   &name));
    kernel_name_ = name;
    // Destroy the program.
    NVRTC_CALL(hiprtcDestroyProgram(&program));
    int device;
    hipDevice_t cuDevice;
    hipCtx_t context;
    hipModule_t module;
    CUDA_CALL(hipGetDevice(&device));
    CUDA_DRIVER_CALL(hipDeviceGet(&cuDevice, device));
    CUDA_DRIVER_CALL(hipDevicePrimaryCtxRetain(&context, cuDevice));
    CUDA_DRIVER_CALL(hipModuleLoadData(&module, &ptx_[0]));
    CUDA_DRIVER_CALL(hipModuleGetFunction(&kernel_,
                                         module,
                                         kernel_name_.c_str()));
    initialized_ = true;
  }
  Stream<gpu>* s = ctx.get_stream<gpu>();
  auto stream = Stream<gpu>::GetStream(s);
  std::vector<void*> args;
  size_t N = (outputs[0].shape_.Size() + nvec - 1)/nvec;
  args.push_back(&N);

  unsigned int num_blocks = (N + FusedOp::NTHREADS - 1) / FusedOp::NTHREADS;
  std::vector<void*> ptrs;
  std::vector<std::vector<int>> strides;
  for (const auto &data : inputs) {
    MSHADOW_TYPE_SWITCH(data.type_flag_, DType, {
      int ndim = data.ndim();
      Tensor<gpu, 1, DType> tensor = data.FlatTo1D<gpu, DType>(s);
      ptrs.push_back(tensor.dptr_);
      strides.push_back(std::vector<int>(ndim));
      std::vector<int>& tensor_strides = strides.back();
      tensor_strides[ndim-1] = 1;
      for (int i = ndim-2; i >= 0; i--) {
        tensor_strides[i] = tensor_strides[i+1] * data.shape_[i+1];
      }
    });
  }
  for (const auto &data : outputs) {
    MSHADOW_TYPE_SWITCH(data.type_flag_, DType, {
      Tensor<gpu, 1, DType> tensor = data.FlatTo1D<gpu, DType>(s);
      ptrs.push_back(tensor.dptr_);
    });
  }
  for (auto &tensor_strides : strides) {
    args.push_back(tensor_strides.data());
  }
  for (auto &ptr : ptrs) {
    args.push_back(reinterpret_cast<void *>(&ptr));
  }
  CUDA_DRIVER_CALL(
      hipModuleLaunchKernel(kernel_,
        num_blocks, 1, 1,          // grid dim
        FusedOp::NTHREADS, 1, 1,   // block dim
        0, stream,                 // shared mem and stream
        &(args[0]), 0));           // arguments
}

void FusedOpForwardGPU(const nnvm::NodeAttrs& attrs,
                    const OpContext &ctx,
                    const std::vector<TBlob> &inputs,
                    const std::vector<OpReqType> &req,
                    const std::vector<TBlob> &outputs) {
  const FusedOpPtr& op = nnvm::get<FusedOpPtr>(attrs.parsed);
  op->Forward<gpu>(attrs, ctx, inputs, req, outputs);
}

NNVM_REGISTER_OP(_FusedOp)
.set_attr<FCompute>("FCompute<gpu>", FusedOpForwardGPU);

}  // namespace mxnet
