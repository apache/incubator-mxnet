#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

#include <sys/stat.h>
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <nnvm/pass_functions.h>
#include <algorithm>
#include <mutex>
#include "./fused_op.h"
#include "./fused_op-inl.h"
#include "../operator_common.h"
#include "../elemwise_op_common.h"
#include "../../executor/exec_pass.h"
#include "../../common/cuda_utils.h"

namespace mxnet {

namespace {

inline std::string mshadowTypeToString(int type) {
  switch (type) {
    case mshadow::kFloat32:
      return "float";
    case mshadow::kFloat64:
      return "double";
    case mshadow::kFloat16:
      return "half";
    case mshadow::kUint8:
      return "unsigned char";
    case mshadow::kInt8:
      return "char";
    case mshadow::kInt32:
      return "int";
    case mshadow::kInt64:
      return "long long";
    default:
      LOG(FATAL) << "Unknown type enum " << type;
  }
  return "";
}

inline int mshadowTypeToVectorLength(int type) {
  switch (type) {
    case mshadow::kFloat32:
      return 1;
    case mshadow::kFloat64:
      return 1;
    case mshadow::kFloat16:
      return 2;
    case mshadow::kUint8:
      return 4;
    case mshadow::kInt8:
      return 4;
    case mshadow::kInt32:
      return 1;
    case mshadow::kInt64:
      return 1;
    default:
      LOG(FATAL) << "Unknown type enum " << type;
  }
  return 0;
}

inline void replaceString(std::string *input, const std::string old, const std::string repl) {
    size_t pos = 0;
    while ((pos = input->find(old, pos)) != std::string::npos) {
        input->replace(pos, old.size(), repl);
        pos += repl.size();
    }
}

std::string ParseOpDescription(const std::vector<std::string>& op_desc,
                               const std::map<std::pair<int, int>, std::string>& variables,
                               const nnvm::IndexedGraph::Node& node) {
  const auto* source = node.source;
  std::string fmt = op_desc[0];
  for (size_t j = 1; j < op_desc.size(); ++j) {
    const std::string& desc = op_desc[j];
    std::string sub;
    if (desc[0] == '_') {
      // Argument
      const int arg_id = std::stoi(desc.substr(1));
      sub = variables.at({node.inputs[arg_id].node_id, node.inputs[arg_id].index});
    } else {
      sub = source->attrs.dict.at(desc);
    }
    size_t pos = fmt.find("%");
    CHECK_NE(pos, std::string::npos);
    fmt.replace(pos, 1, sub);
  }
  return fmt;
}

}  // namespace

void FusedOp::GenerateCode(const std::vector<OpReqType> &req,
                           const std::vector<int> &in_dtypes,
                           const std::vector<int> &out_dtypes,
                           const std::vector<int> &in_ndims,
                           const int nvec,
                           const std::string &kernel_name) {
  const auto& g = this->symbol_.indexed_graph();
  std::string code = "";
  int temp_name_counter = 0;
  using NodeEntry = nnvm::IndexedGraph::NodeEntry;
  std::map<std::pair<int, int>, std::string> variables;
  std::map<int, int> load_index;

  std::vector<uint32_t> outputs(g.num_nodes());

  for (size_t i = 0; i < g.num_nodes(); ++i) {
    const auto& node = g[i];
    if (node.source != nullptr) {
      outputs[i] = node.source->num_outputs();
    } else {
      outputs[i] = 0;
    }
  }

  for (size_t i = 0; i < g.num_nodes(); ++i) {
    const auto& node = g[i];
    const auto* source = node.source;
    if (source != nullptr) {
        if (source->is_variable()) {
            load_index[i] = 1;
        } else {
            std::string op_name = source->op()->name;
            if (fusion::slice_ops.find(op_name) != fusion::slice_ops.end()) {
                load_index[node.inputs[0].node_id] = 0;
            }
        }
    }
  }
  for (size_t i = 0; i < g.num_nodes(); ++i) {
    const auto& node = g[i];
    const auto* source = node.source;
    if (source != nullptr) {
      if (source->is_variable()) {
        if (load_index[i]) {
          const auto& var_name = source->attrs.name;
          code += "const auto vec_" + var_name + " = load_index<nvec>(" +
                   var_name + ", offset);\n";
          variables[{i, 0}] = var_name;
        }
        CHECK_EQ(outputs[i], 1);
      } else {
        std::string op_name = source->op()->name;
        if (fusion::slice_ops.find(op_name) != fusion::slice_ops.end()) {
          int arg_id = node.inputs[0].node_id;
          const auto& var_name = g[arg_id].source->attrs.name;
          const auto vec_name = "vec_" + var_name + "_" + std::to_string(i);
          load_index[arg_id] = 0;
          auto parse_tuple = [](const std::string& input, const std::string def) {
            std::string out = input;
            replaceString(&out, "(", "{");
            replaceString(&out, ")", "}");
            replaceString(&out, "None", def);
            return out;
          };
          std::string begin = parse_tuple(source->attrs.dict.at("begin"), "0");
          std::string end = parse_tuple(source->attrs.dict.at("end"), "INT_MAX");
          if (op_name == "slice_axis") {
            std::string axis = source->attrs.dict.at("axis");
            std::string begin_var_name = var_name + "_" + std::to_string(i) + "_begin";
            std::string end_var_name = var_name + "_" + std::to_string(i) + "_end";
            code += "Shape<ndim_" + var_name + "> "+ begin_var_name + ";\n";
            code += "Shape<ndim_" + var_name + "> "+ end_var_name + ";\n";
            code += begin_var_name + ".set(0);\n";
            code += end_var_name + ".set(INT_MAX);\n";
            code += begin_var_name + "["+axis+"] = " + begin + ";\n";
            code += end_var_name + "["+axis+"] = " + end + ";\n";
            begin = begin_var_name;
            end = end_var_name;
          }
          code += "const auto " + vec_name + " = load_slice<nvec>(" +
                  var_name + ", " + var_name + "_shape," + begin +
                  "," + end + ", offset);\n";
          CHECK_EQ(outputs[i], 1);
          variables[{i, 0}] = vec_name;
          continue;
        }
      }
    }
  }

  size_t counter = 0;
  for (const auto& entry : g.outputs()) {
    std::string var_name = "output" + std::to_string(counter);
    code += "VectorType<DType_" + var_name + \
            ", nvec> vec_" + var_name + ";\n";
    ++counter;
  }

  code += "for (int j = 0; j < nvec; j++ ) {\n";


  for (size_t i = 0; i < g.num_nodes(); ++i) {
    const auto& node = g[i];
    const auto* source = node.source;
    if (source != nullptr) {
      std::string var_name = "temp" + std::to_string(temp_name_counter++);
      if (source->is_variable()) {
        if (load_index[i]) {
            code += "const auto " + var_name + " = load(vec_" + variables[{i, 0}] + ".x[j]);\n";
            CHECK_EQ(outputs[i], 1);
            variables[{i, 0}] = var_name;
        }
      } else {
        std::string op_name = source->op()->name;
        if (fusion::ops_desc.find(op_name) != fusion::ops_desc.end()) {
          const std::vector<std::vector<std::string>>& op_descs =
            fusion::ops_desc.at(op_name);
          CHECK_EQ(outputs[i], op_descs.size());
          size_t count = 0;
          for (const auto& op_desc : op_descs) {
            var_name = "temp" + std::to_string(temp_name_counter++);
            const std::string& fmt = ParseOpDescription(op_desc, variables, node);
            code += "const auto " + var_name + " = " + fmt + ";\n";
            variables[{i, count}] = var_name;
            ++count;
          }
          continue;
        }

        if (fusion::slice_ops.find(op_name) != fusion::slice_ops.end()) {
          code += "const auto " + var_name + " = load(" + variables[{i, 0}] + ".x[j]);\n";
          variables[{i, 0}] = var_name;
          continue;
        }


        // Special cases with variable number
        // of inputs/outputs, listed in
        // fusion::variable_io_ops
        if (op_name == "add_n") {
          CHECK_EQ(outputs[i], 1);
          const auto& arg = variables[{node.inputs[0].node_id, node.inputs[0].index}];
          code += "auto " + var_name + " = " + arg + ";\n";
          for (size_t inp = 1; inp < node.inputs.size(); ++inp) {
            const auto& temp_arg = variables[{node.inputs[inp].node_id, node.inputs[inp].index}];
            code += var_name + " = add(" + var_name + ", " + temp_arg + ");\n";
          }
          variables[{i, 0}] = var_name;
          continue;
        }

        if (op_name == "_backward_Activation") {
          CHECK_EQ(outputs[i], 1);
          std::string act_type = node.source->attrs.dict.at("act_type");
          std::string rhs, lhs;
          rhs = variables[{node.inputs[0].node_id, node.inputs[0].index}];
          if (act_type == "relu" ||
              act_type == "sigmoid" ||
              act_type == "tanh") {
            lhs = variables[{node.inputs[1].node_id, node.inputs[1].index}];
          } else {
            lhs = variables[{node.inputs[2].node_id, node.inputs[2].index}];
          }
          code += "const auto " + var_name + " = backward_" + act_type +
                  "(" + lhs + ", " + rhs + ");\n";

          variables[{i, 0}] = var_name;
          continue;
        }

        if (op_name == "amp_multicast" || op_name == "_backward_amp_multicast") {
          CHECK_EQ(outputs[i], node.inputs.size());
          for (size_t counter = 0; counter < outputs[i]; ++counter) {
            const auto& input = node.inputs[counter];
            var_name = "temp" + std::to_string(temp_name_counter++);
            const auto& arg = variables[{input.node_id, input.index}];
            code += "const auto " + var_name + " = " + arg + ";\n";
            variables[{i, counter}] = var_name;
          }
          continue;
        }

        LOG(FATAL) << "Unrecognized op " + op_name;
      }
    } else {
      LOG(FATAL) << "Encountered node with NULL source.";
    }
  }

  counter = 0;
  for (const auto& entry : g.outputs()) {
    const std::string& var = variables[{entry.node_id, entry.index}];
    const auto var_name = "output" + std::to_string(counter);
    code += "vec_" + var_name + ".x[j] = store("+ var +", " + var_name + ");\n";
    ++counter;
  }

  code += "}\n";

  counter = 0;

  for (const auto& entry : g.outputs()) {
    const std::string& var = variables[{entry.node_id, entry.index}];
    if (req[counter] == kWriteTo || req[counter] == kWriteInplace) {
      const auto var_name = "output" + std::to_string(counter);
      code += "store_index(vec_" + var_name + ", i, " + var_name + ");\n";
    } else if (req[counter] == kAddTo) {
      const auto var_name = "output" + std::to_string(counter);
      code += "store_add_index(vec_" + var_name + ", i, " + var_name + ");\n";
    } else if (req[counter] == kNullOp) {
      // NULL req, do not do anything
    } else {
      LOG(FATAL) << "Encountered unexpected req.";
    }
    ++counter;
  }

  this->code_ = code;

  // Add boilerplate and type information
  if (dmlc::GetEnv("MXNET_FUSION_VERBOSE", false)) {
    LOG(INFO) << code_;
  }
  std::string kernel_params = "";
  std::string tensor_params = "";
  nnvm::Symbol sym;
  sym.outputs = this->symbol_.outputs;
  const std::vector<std::string> input_names = sym.ListInputNames(nnvm::Symbol::kAll);
  size_t num_params = in_dtypes.size() + out_dtypes.size();
  size_t i = 0;
  std::string aux_code = "static const int nvec = " + std::to_string(nvec) + ";\n";
  for (const auto &type : in_dtypes) {
    std::string type_name = mshadowTypeToString(type);
    std::string dtype_var = "DType_" + input_names[i];
    std::string dim_var = "ndim_" + input_names[i];
    aux_code = "using " + dtype_var + " = " + type_name + ";\n" + aux_code;
    aux_code = "static const int " + dim_var + " = " + \
                std::to_string(in_ndims[i]) + ";\n" + aux_code;
    tensor_params += dtype_var + "* " +input_names[i];
    kernel_params += " const Shape<" + dim_var + "> " + input_names[i]+"_shape";
    ++i;
    if (i < num_params) {
      tensor_params += ", ";
      kernel_params += ", ";
    }
  }
  for (const auto &type : out_dtypes) {
    std::string type_name = mshadowTypeToString(type);
    std::string out_name = "output" + std::to_string(i - in_dtypes.size());
    std::string dtype_var = "DType_" + out_name;
    aux_code = "using " + dtype_var + " = " + type_name + ";\n" + aux_code;
    tensor_params += dtype_var + "* " + out_name;
    ++i;
    if (i < num_params) {
      tensor_params += ", ";
    }
  }
  kernel_params += tensor_params;
  code_ = std::string(fusion::fp16_support_string) + "\n" +
          fusion::type_support_string + "\n" +
          fusion::function_definitions + "\n" +
          aux_code + "\n" +
          "__global__ void FusedKernel_" + kernel_name +
          "(size_t N, " + kernel_params + ") {\n" +
          fusion::kernel_begin + "\n" +
          code_ + "\n" +
          fusion::kernel_end;
}

void FusedOp::CompileCode(const std::string &kernel_name) {
  // Guard NVRTC calls
  std::lock_guard<std::mutex> lock_nvrtc(mutex_);
  hiprtcProgram program;
  NVRTC_CALL(
      hiprtcCreateProgram(&program,                                  // prog
                         &code_[0],                                 // buffer
                         (kernel_name + "_kernel.cu").c_str(),      // name
                         0,                                         // numHeaders
                         NULL,                                      // headers
                         NULL));                                    // includeNames
  std::string gpu_arch = "--gpu-architecture=compute_" +
                         std::to_string(this->cc_major_) +
                         std::to_string(this->cc_minor_);

  const char *opts[] = {gpu_arch.c_str(),
                        "--std=c++11",
                        "-default-device"};
  const std::string kernel_name_demangled = "FusedKernel_" + kernel_name;
  NVRTC_CALL(hiprtcAddNameExpression(program, (kernel_name_demangled).c_str()));

  hiprtcResult compileResult = hiprtcCompileProgram(program,  // prog
                                                  3,        // numOptions
                                                  opts);    // options
  // Obtain compilation log from the program.
  size_t logSize;
  NVRTC_CALL(hiprtcGetProgramLogSize(program, &logSize));
  std::string log(logSize, '\0');
  NVRTC_CALL(hiprtcGetProgramLog(program, &log[0]));
  CHECK_EQ(compileResult, HIPRTC_SUCCESS) << "NVRTC Compilation failed.\n" << log;
  // Obtain PTX from the program.
  size_t ptxSize;
  NVRTC_CALL(hiprtcGetCodeSize(program, &ptxSize));
  ptx_.reserve(ptxSize);
  NVRTC_CALL(hiprtcGetCode(program, &ptx_[0]));
  const char *name;
  NVRTC_CALL(hiprtcGetLoweredName(program,
                                 kernel_name_demangled.c_str(),
                                 &name));
  kernel_name_ = name;
  // Destroy the program.
  NVRTC_CALL(hiprtcDestroyProgram(&program));
  int device;
  hipDevice_t cuDevice;
  hipCtx_t context;
  hipModule_t module;
  CUDA_CALL(hipGetDevice(&device));
  CUDA_DRIVER_CALL(hipDeviceGet(&cuDevice, device));
  CUDA_DRIVER_CALL(hipDevicePrimaryCtxRetain(&context, cuDevice));
  CUDA_DRIVER_CALL(hipModuleLoadData(&module, &ptx_[0]));
  CUDA_DRIVER_CALL(hipModuleGetFunction(&kernel_,
                                       module,
                                       kernel_name_.c_str()));
}

bool FusedOp::CheckComputeCapability(const OpContext &ctx) {
  const int dev_id = ctx.run_ctx.ctx.dev_id;
  const int cc_major = ComputeCapabilityMajor(dev_id);
  const int cc_minor = ComputeCapabilityMinor(dev_id);

  const bool ret = cc_major == this->cc_major_ && cc_minor == this->cc_minor_;
  this->cc_major_ = cc_major;
  this->cc_minor_ = cc_minor;
  return ret;
}

template <>
void FusedOp::Forward<gpu>(const nnvm::NodeAttrs& attrs,
                           const OpContext &ctx,
                           const std::vector<TBlob> &inputs,
                           const std::vector<OpReqType> &req,
                           const std::vector<TBlob> &outputs) {
  using namespace mshadow;
  std::lock_guard<std::mutex> lock(my_mutex_);
  CHECK_GE(outputs.size(), 1) << "There needs to be at least 1 output.";

  std::vector<int> in_dtypes;
  std::vector<int> in_ndims;
  std::vector<int> out_dtypes;
  int ndim = outputs[0].ndim();
  int nvec = 1;

  CHECK_EQ(inputs.size(), inputs_.size());
  for (size_t counter = 0; counter < inputs.size(); ++counter) {
    const auto& blob = inputs[counter];
    in_dtypes.push_back(blob.type_flag_);
    in_ndims.push_back(blob.ndim());
    initialized_ = initialized_ && (blob.type_flag_ == inputs_[counter].dtype);
    inputs_[counter].dtype = blob.type_flag_;
    nvec = max(nvec, mshadowTypeToVectorLength(blob.type_flag_));
  }

  CHECK_EQ(outputs.size(), outputs_.size());
  for (size_t counter = 0; counter < outputs.size(); ++counter) {
    const auto& blob = outputs[counter];
    out_dtypes.push_back(blob.type_flag_);
    initialized_ = initialized_ && (blob.type_flag_ == outputs_[counter].dtype);
    outputs_[counter].dtype = blob.type_flag_;
    nvec = max(nvec, mshadowTypeToVectorLength(blob.type_flag_));
  }

  // Check and save compute capability of the current GPU
  if (!CheckComputeCapability(ctx)) initialized_ = false;

  initialized_ = initialized_ && (req == saved_reqs_);
  saved_reqs_ = req;

  if (!initialized_) {
    this->GenerateCode(req, in_dtypes, out_dtypes, in_ndims, nvec, attrs.name);
    this->CompileCode(attrs.name);
    initialized_ = true;
  }
  Stream<gpu>* s = ctx.get_stream<gpu>();
  auto stream = Stream<gpu>::GetStream(s);
  std::vector<void*> args;
  size_t N = (outputs[0].shape_.Size() + nvec - 1)/nvec;
  args.push_back(&N);

  unsigned int num_blocks = (N + FusedOp::NTHREADS - 1) / FusedOp::NTHREADS;
  std::vector<void*> ptrs;
  std::vector<std::vector<int>> shapes;
  for (const auto &data : inputs) {
    MSHADOW_TYPE_SWITCH(data.type_flag_, DType, {
      int ndim = data.ndim();
      Tensor<gpu, 1, DType> tensor = data.FlatTo1D<gpu, DType>(s);
      ptrs.push_back(tensor.dptr_);
      shapes.push_back(std::vector<int>(ndim));
      std::vector<int>& tensor_shapes = shapes.back();
      for (int i = ndim-1; i >= 0; i--) {
        tensor_shapes[i] = data.shape_[i];
      }
    });
  }
  for (const auto &data : outputs) {
    MSHADOW_TYPE_SWITCH(data.type_flag_, DType, {
      Tensor<gpu, 1, DType> tensor = data.FlatTo1D<gpu, DType>(s);
      ptrs.push_back(tensor.dptr_);
    });
  }
  for (auto &tensor_shapes : shapes) {
    args.push_back(tensor_shapes.data());
  }
  for (auto &ptr : ptrs) {
    args.push_back(reinterpret_cast<void *>(&ptr));
  }
  CUDA_DRIVER_CALL(
      hipModuleLaunchKernel(kernel_,
        num_blocks, 1, 1,          // grid dim
        FusedOp::NTHREADS, 1, 1,   // block dim
        0, stream,                 // shared mem and stream
        &(args[0]), 0));           // arguments
}

void FusedOpForwardGPU(const nnvm::NodeAttrs& attrs,
                    const OpContext &ctx,
                    const std::vector<TBlob> &inputs,
                    const std::vector<OpReqType> &req,
                    const std::vector<TBlob> &outputs) {
  const FusedOpPtr& op = nnvm::get<FusedOpPtr>(attrs.parsed);
  op->Forward<gpu>(attrs, ctx, inputs, req, outputs);
}

NNVM_REGISTER_OP(_FusedOp)
.set_attr<FCompute>("FCompute<gpu>", FusedOpForwardGPU);

}  // namespace mxnet
