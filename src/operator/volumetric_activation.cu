#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2015 by Contributors
 * \file activation.cu
 * \brief
 * \author Bing Xu
*/
#include "./volumetric_activation-inl.h"
#include "./mshadow_op.h"
#if MXNET_USE_CUDNN == 1
#include "./volumetric_cudnn_activation-inl.h"
#endif

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(VolumetricActivationParam param) {
#if MXNET_USE_CUDNN == 1
  return new CuDNNVolumetricActivationOp(param);
#else
  switch(param.act_type) {
    case activation::kReLU:
      return new VolumetricActivationOp<gpu, mshadow_op::relu, mshadow_op::relu_grad>();
    case activation::kSigmoid:
      return new VolumetricActivationOp<gpu, mshadow_op::sigmoid, mshadow_op::sigmoid_grad>();
    case activation::kTanh:
      return new VolumetricActivationOp<gpu, mshadow_op::tanh, mshadow_op::tanh_grad>();
    default:
      LOG(FATAL) << "unknown activation";
      return NULL;
  }
#endif  // MXNET_USE_CUDNN
}
}  // op
}  // namespace mxnet

