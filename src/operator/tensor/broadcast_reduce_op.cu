#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2016 by Contributors
 * \file broadcast_reduce_op.cu
 * \brief GPU Implementation of broadcast and reduce functions.
 */
#include "./broadcast_reduce_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(sum)
.attr<FCompute>("FCompute<gpu>", ReduceAxesCompute<gpu, mshadow::red::sum>);

NNVM_REGISTER_OP(_backward_sum)
.attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseNone<gpu>);

NNVM_REGISTER_OP(max)
.attr<FCompute>("FCompute<gpu>", ReduceAxesCompute<gpu, mshadow::red::maximum>);

NNVM_REGISTER_OP(_backward_max)
.attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseInOut<gpu, mshadow_op::eq>);

NNVM_REGISTER_OP(min)
.attr<FCompute>("FCompute<gpu>", ReduceAxesCompute<gpu, mshadow::red::minimum>);

NNVM_REGISTER_OP(_backward_min)
.attr<FCompute>("FCompute<gpu>", ReduceAxesBackwardUseInOut<gpu, mshadow_op::eq>);

NNVM_REGISTER_OP(broadcast_axis)
.attr<FCompute>("FCompute<gpu>", BroadcastCompute<gpu>);

NNVM_REGISTER_OP(broadcast_to)
.attr<FCompute>("FCompute<gpu>", BroadcastCompute<gpu>);

}  // namespace op
}  // namespace mxnet
