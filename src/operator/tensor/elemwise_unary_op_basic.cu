#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2016 by Contributors
 * \file elemwise_unary_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {
// copy
NNVM_REGISTER_OP(_copy)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

NNVM_REGISTER_OP(_backward_copy)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

// identity output as first input, but attributes are constrainted to be like rhs
NNVM_REGISTER_OP(_identity_attr_like_rhs)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

// negative
NNVM_REGISTER_OP(negative)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::negation>);

// abs
NNVM_REGISTER_OP(abs)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::abs>);

NNVM_REGISTER_OP(_backward_abs)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sign> >);

// sign
NNVM_REGISTER_OP(sign)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::sign>);

NNVM_REGISTER_OP(_backward_sign)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sign_grad> >);

// round
NNVM_REGISTER_OP(round)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::round>);

// ceil
NNVM_REGISTER_OP(ceil)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::ceil>);

// floor
NNVM_REGISTER_OP(floor)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::floor>);

// square
NNVM_REGISTER_OP(square)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::square>);

NNVM_REGISTER_OP(_backward_square)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::square_grad> >);

// sqrt
NNVM_REGISTER_OP(sqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::square_root>);

NNVM_REGISTER_OP(_backward_sqrt)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::square_root_grad> >);

// rsqrt
NNVM_REGISTER_OP(rsqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::reciprocal_square_root>);

NNVM_REGISTER_OP(_backward_rsqrt)
.set_attr<FCompute>("FCompute<gpu>",
  BinaryCompute<gpu, unary_bwd<mshadow_op::reciprocal_square_root_grad> >);

// exp
NNVM_REGISTER_OP(exp)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::exp>);

// log
NNVM_REGISTER_OP(log)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::log>);

NNVM_REGISTER_OP(_backward_log)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::log_grad> >);

// log1p
NNVM_REGISTER_OP(log1p)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::log1p>);

NNVM_REGISTER_OP(_backward_log1p)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::log1p_grad> >);

// expm1
NNVM_REGISTER_OP(expm1)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::expm1>);

NNVM_REGISTER_OP(_backward_expm1)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::exp> >);

}  // namespace op
}  // namespace mxnet

