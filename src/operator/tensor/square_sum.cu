/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file square_sum.cu
 * \brief GPU Implementation of square_sum op.
 */
#include "./square_sum-inl.h"

namespace mxnet {
namespace op {

template<>
void CheckSameIdx<gpu>(mshadow::Stream<gpu>* s,
                       const TBlob ograd_row_idx,
                       const TBlob in_row_idx) {
  MSHADOW_IDX_TYPE_SWITCH(ograd_row_idx.type_flag_, IType, {
    const IType* ograd_idx = ograd_row_idx.dptr<IType>();
    const IType* in_idx = in_row_idx.dptr<IType>();
    const nnvm::dim_t idx_size = ograd_row_idx.Size();
    int32_t is_same = 0;
    int32_t* is_same_ptr = NULL;
    CUDA_CALL(hipMalloc(&is_same_ptr, sizeof(int32_t)));
    mxnet_op::Kernel<mxnet_op::set_zero, gpu>::Launch(s, 1, is_same_ptr);
    mxnet_op::Kernel<CheckSameIdxKernel, gpu>::Launch(s, idx_size, ograd_idx, in_idx, &is_same);
    CUDA_CALL(hipMemcpy(&is_same, is_same_ptr, sizeof(int32_t), hipMemcpyDeviceToHost));
    CHECK_EQ(is_same, 0) << "SquareSumRspGradImpl only supports"
                            " equal ograd_row_idx and input_row_idx"
                            " when ograd and input are both"
                            " row-sparse and input data is not a full"
                            " row-sparse matrix";
    CUDA_CALL(hipFree(is_same_ptr));
  })
}


NNVM_REGISTER_OP(_square_sum)
.set_attr<FComputeEx>("FComputeEx<gpu>", SquareSumOpForwardEx<gpu>);

NNVM_REGISTER_OP(_backward_square_sum)
.set_attr<FComputeEx>("FComputeEx<gpu>", SquareSumOpBackwardEx<gpu>);

}  // namespace op
}  // namespace mxnet
