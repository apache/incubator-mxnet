#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2016 by Contributors
 * \file elemwise_unary_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {
// sin
NNVM_REGISTER_OP(sin)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::sin>);

NNVM_REGISTER_OP(_backward_sin)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sin_grad> >);

// cos
NNVM_REGISTER_OP(cos)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::cos>);

NNVM_REGISTER_OP(_backward_cos)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::cos_grad> >);

// tan
NNVM_REGISTER_OP(tan)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::tan>);

NNVM_REGISTER_OP(_backward_tan)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::tan_grad> >);

// arcsin
NNVM_REGISTER_OP(arcsin)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arcsin>);

NNVM_REGISTER_OP(_backward_arcsin)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arcsin_grad> >);

// arccos
NNVM_REGISTER_OP(arccos)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arccos>);

NNVM_REGISTER_OP(_backward_arccos)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arccos_grad> >);

// arctan
NNVM_REGISTER_OP(arctan)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arctan>);

NNVM_REGISTER_OP(_backward_arctan)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arctan_grad> >);

// degrees
NNVM_REGISTER_OP(degrees)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::degrees>);

NNVM_REGISTER_OP(_backward_degrees)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::degrees_grad> >);

// cosh
NNVM_REGISTER_OP(cosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::cosh>);

NNVM_REGISTER_OP(_backward_cosh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::cosh_grad> >);

// sinh
NNVM_REGISTER_OP(sinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::sinh>);

NNVM_REGISTER_OP(_backward_sinh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::sinh_grad> >);

// tanh
NNVM_REGISTER_OP(tanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::tanh>);

NNVM_REGISTER_OP(_backward_tanh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::tanh_grad> >);

// arcsinh
NNVM_REGISTER_OP(arcsinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arcsinh>);

NNVM_REGISTER_OP(_backward_arcsinh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arcsinh_grad> >);

// arccosh
NNVM_REGISTER_OP(arccosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arccosh>);

NNVM_REGISTER_OP(_backward_arccosh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arccosh_grad> >);

// arctanh
NNVM_REGISTER_OP(arctanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::arctanh>);

NNVM_REGISTER_OP(_backward_arctanh)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::arctanh_grad> >);

// gamma
NNVM_REGISTER_OP(gamma)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::gamma>);

NNVM_REGISTER_OP(_backward_gamma)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::gamma_grad> >);

// gammaln
NNVM_REGISTER_OP(gammaln)
.set_attr<FCompute>("FCompute<gpu>", UnaryCompute<gpu, mshadow_op::gammaln>);

NNVM_REGISTER_OP(_backward_gammaln)
.set_attr<FCompute>("FCompute<gpu>", BinaryCompute<gpu, unary_bwd<mshadow_op::gammaln_grad> >);

}  // namespace op
}  // namespace mxnet
