#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file indexing_op.cu
 * \brief
 * \author Siyi Li, Chi Zhang
*/

#include "./indexing_op.h"
#include "./util/tensor_util-inl.cuh"
#include <chrono>

namespace mxnet {
namespace op {

/*! \brief If there are out-of-bound indices, out will be assigned to 1.
 */

struct is_valid_check {
  template<typename DType>
  MSHADOW_XINLINE static void Map(int i, int32_t* out, const DType* data,
                                  const DType min, const DType max) {
    if (data[i] < min || data[i] > max) *out = 1;
  }
};


struct AddTakeGradRspGPUKernel {
  template<typename DType, typename IType>
  __device__ __forceinline__ static void Map(int tid,
                                             DType* out,
                                             const nnvm::dim_t* prefix_sum,
                                             const IType* data,
                                             const DType* ograd,
                                             const nnvm::dim_t row_length) {
    using nnvm::dim_t;
    const dim_t data_i = tid / row_length;
    const dim_t grad_i = tid % row_length;
    const dim_t irow = static_cast<dim_t>(data[data_i]);
    const dim_t rsp_row = prefix_sum[irow] - 1;
    const DType val = ograd[data_i * row_length + grad_i];
    atomicAdd(static_cast<DType *>(&(out[rsp_row*row_length+grad_i])), val);
  }
};

template<>
void SparseEmbeddingOpForwardRspImpl<gpu>(const OpContext& ctx,
                                          const TBlob& data,
                                          const NDArray& weight,
                                          const OpReqType req,
                                          const TBlob& output) {
  if (req == kNullOp) return;
  using namespace rowsparse;
  using namespace mxnet_op;
  mshadow::Stream<gpu>* s = ctx.get_stream<gpu>();
  // zeros weight
  if (req == kWriteTo && !weight.storage_initialized()) {
    size_t out_size = output.shape_.Size();
    MSHADOW_TYPE_SWITCH(output.type_flag_, DType, {
      Fill<false>(s, TBlob(output.dptr<DType>(), mshadow::Shape1(out_size),
          gpu::kDevMask), kWriteTo, 0);
    })
    return;
  }
  // check out-of-bound indices
  int32_t is_valid = 0;
  MSHADOW_TYPE_SWITCH(data.type_flag_, DType, {
    DType min = 0;
    DType max = static_cast<DType>(weight.shape()[0] - 1);
    DType* data_ptr = data.dptr<DType>();
    size_t data_size = data.shape_.Size();
    Tensor<gpu, 1, char> workspace = ctx.requested[0]
        .get_space_typed<gpu, 1, char>(Shape1(sizeof(int32_t)), s);
    int32_t* is_valid_ptr = reinterpret_cast<int32_t*>(workspace.dptr_);
    Kernel<set_zero, gpu>::Launch(s, 1, is_valid_ptr);
    Kernel<is_valid_check, gpu>::Launch(s, data_size, is_valid_ptr, data_ptr, min, max);
    CUDA_CALL(hipMemcpy(&is_valid, is_valid_ptr, sizeof(int32_t),
              hipMemcpyDeviceToHost));
  })
  CHECK_EQ(is_valid, 0) << "SparseEmbedding input contains data out of bound";
  // the weight is actually dense
  if (weight.aux_shape(kIdx)[0] == weight.shape()[0]) {
    EmbeddingOpForwardDnsImpl<gpu>(s, data, weight.data(), req, output);
  } else {
    EmbeddingOpForwardRspImpl<gpu>(s, data, weight, req, output);
  }
}

inline void SparseEmbeddingOpBackwardRspImpl3(const OpContext& ctx,
                                              const TBlob& ograd,
                                              const TBlob& data,
                                              const OpReqType req,
                                              const NDArray& output) {
  using namespace mshadow;
  using namespace mxnet_op;
  using namespace mshadow::expr;
  using namespace rowsparse;
  using nnvm::dim_t;
  if (req == kNullOp) return;
  CHECK_EQ(req, kWriteTo) << "SparseEmbedding layer doesn't support "
                          << "weight gradient calculation with req != write";

  // Request temporary storage for marking non-zero rows and prefix sum
  Stream<gpu> *s = ctx.get_stream<gpu>();
  dim_t num_rows = output.shape()[0];
  dim_t row_length = output.shape()[1];
  dim_t data_size = static_cast<dim_t>(data.shape_.Size());
  dim_t num_threads;
  if (data_size == 0) {
    FillZerosRspImpl(s, output);
    return;
  }
   s->Wait();
  auto t0 = std::chrono::duration_cast<std::chrono::microseconds>(
    std::chrono::high_resolution_clock::now().time_since_epoch()).count();

  MSHADOW_TYPE_SWITCH(data.type_flag_, IType, {
    MSHADOW_TYPE_SWITCH(ograd.type_flag_, DType, {
      MSHADOW_IDX_TYPE_SWITCH(output.aux_type(kIdx), RType, {
        dim_t* prefix_sum = NULL;
        void* temp_storage = NULL;
        dim_t* sorted_data = NULL;
        dim_t* original_idx = NULL;
        // calculate resource bytes
        size_t row_flg_storage_bytes = num_rows * sizeof(dim_t);
        size_t sorted_data_storage_bytes = data_size * sizeof(dim_t);
        size_t original_idx_storage_bytes = data_size * sizeof(dim_t);
        size_t sum_workspace_bytes = 0;
        size_t sort_workspace_size = SortByKeyWorkspaceSize<dim_t, dim_t, gpu>(data_size);
        hipcub::DeviceScan::InclusiveSum(temp_storage,
                                      sum_workspace_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      Stream<gpu>::GetStream(s));
        // temp_workspace is shared by inclusive sum and sort
        size_t temp_workspace_bytes = std::max(sum_workspace_bytes, sort_workspace_size);
        size_t total_storage_bytes = row_flg_storage_bytes + sorted_data_storage_bytes +
                                     original_idx_storage_bytes + temp_workspace_bytes;

        // request resource and split it. layout =
        // row_flg/prefixsum, sorted_data, original_idx, temp_storage
        Tensor<gpu, 1, char> workspace = ctx.requested[0]
            .get_space_typed<gpu, 1, char>(Shape1(total_storage_bytes), s);
        prefix_sum = reinterpret_cast<dim_t*>(workspace.dptr_);
        sorted_data = reinterpret_cast<dim_t*>(workspace.dptr_ + row_flg_storage_bytes);
        original_idx = reinterpret_cast<dim_t*>(workspace.dptr_ + row_flg_storage_bytes +
                                                sorted_data_storage_bytes);
        temp_storage = workspace.dptr_ + total_storage_bytes - temp_workspace_bytes;
        // compute row flags and prefix sum
        num_threads = num_rows;
   s->Wait();
  auto t1 = std::chrono::duration_cast<std::chrono::microseconds>(
    std::chrono::high_resolution_clock::now().time_since_epoch()).count();
        Fill<false>(s, TBlob(prefix_sum, Shape1(num_threads), gpu::kDevMask), kWriteTo, 0);
        Kernel<MarkRowFlgKernel, gpu>::Launch(s, data_size, prefix_sum, data.dptr<IType>());
        hipcub::DeviceScan::InclusiveSum(temp_storage,
                                      temp_workspace_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      mshadow::Stream<gpu>::GetStream(s));
        // retrieve nnr and allocate output
   s->Wait();
  auto t2 = std::chrono::duration_cast<std::chrono::microseconds>(
    std::chrono::high_resolution_clock::now().time_since_epoch()).count();
        dim_t nnr = 0;
        CUDA_CALL(hipMemcpy(&nnr, &prefix_sum[num_rows-1], sizeof(dim_t),
            hipMemcpyDeviceToHost));
        output.CheckAndAlloc({Shape1(nnr)});
        // fill row_idx array of output matrix, using the row_flg values
        RType* grad_row_idx = output.aux_data(kIdx).dptr<RType>();
        Kernel<FillRspRowIdxKernel, gpu>::Launch(s, num_rows,
            grad_row_idx, prefix_sum, num_rows);

        // make a copy of the data, to be sorted
        TBlob sorted_data_blob(sorted_data, Shape1(data_size), gpu::kDevMask);
        auto sorted_data_tensor = sorted_data_blob.FlatTo1D<gpu, dim_t>(s);
        mxnet_op::copy(s, sorted_data_blob, data);

        // generate original idx
        Tensor<gpu, 1, dim_t> original_idx_tensor(original_idx, Shape1(data_size), s);
        Kernel<range_fwd, gpu>::Launch(s, data_size, 1, static_cast<dim_t>(0), static_cast<dim_t>(1),
                                       kWriteTo, original_idx);
        // sort data with its original idx
        int num_bits = ilog2(num_rows - 1);
        char* temp_storage_ptr = reinterpret_cast<char*>(temp_storage);
        Tensor<gpu, 1, char> temp_storage_tensor(temp_storage_ptr,
                                                 Shape1(sort_workspace_size), s);
   s->Wait();
  auto t3 = std::chrono::duration_cast<std::chrono::microseconds>(
    std::chrono::high_resolution_clock::now().time_since_epoch()).count();
        SortByKey(sorted_data_tensor, original_idx_tensor, true,
                  &temp_storage_tensor, 0, num_bits);
   s->Wait();
  auto t4 = std::chrono::duration_cast<std::chrono::microseconds>(
    std::chrono::high_resolution_clock::now().time_since_epoch()).count();

        // accumulate gradients
        DType* grad_data = output.data().dptr<DType>();
        Fill<false>(s, TBlob(grad_data, Shape1(nnr * row_length), gpu::kDevMask),
            kWriteTo, 0);
        //const int SZ = 4;
        //int num_threads_per_data = (row_length + SZ - 1) / SZ;

        //Kernel<AddTakeGradRspGPUKernel2, gpu>::Launch(s, data_size * num_threads_per_data,
        //       grad_data, prefix_sum, sorted_data, data_size, original_idx,
        //       ograd.dptr<DType>(), row_length, num_threads_per_data, SZ);

{
  //using IndexType = IType;
  dim_t* sum_counts_ptr = NULL;
  int* num_runs_ptr = NULL;
  mshadow::Tensor<gpu, 2, DType> dst = output.data().get<gpu, 2, DType>(s);
  hipStream_t stream = mshadow::Stream<gpu>::GetStream(dst.stream_);
  mshadow::Tensor<gpu, 1, dim_t> sorted = sorted_data_tensor;
  mshadow::Tensor<gpu, 1, dim_t> index = original_idx_tensor;
  auto arrshape = ograd.shape_;
  mshadow::Tensor<gpu, 2, DType> src = ograd.get_with_shape<gpu, 2, DType>(
          Shape2(arrshape[0], arrshape.ProdShape(1, arrshape.ndim())), s);

  const int num_unique_est = min(dst.size(0), src.size(0));
  const int max_nthread = 128;
  const int num_y = max(src.size(0)/num_unique_est, 1);
  const int block_dim_x = kWarpSize;
  const int block_dim_y = min(num_y, max_nthread/block_dim_x);
  const int SZ = min((src.size(1) + block_dim_x - 1) / block_dim_x, 4);
  const int grid_dim_x = (src.size(1) + block_dim_x * SZ - 1) / (block_dim_x * SZ);
  const int grid_dim_y = min(num_unique_est, mshadow::cuda::kBaseGridNum);
  dim3 dimBlock(block_dim_x, block_dim_y);
  dim3 dimGrid(grid_dim_x, grid_dim_y);
  // Maximum shared memory usage: 128*4*sizeof(DType), which is 4K for 64bit DType elements
  int shmem_size = dimBlock.x*SZ*dimBlock.y*sizeof(DType);

  CHECK_EQ(dst.size(1), src.size(1)) << "AddTakeGradLargeBatch: shape mismatch";
  CHECK_EQ(index.size(0), src.size(0)) << "AddTakeGradLargeBatch: shape mismatch";
  mshadow::cuda::CheckLaunchParam(dimGrid, dimBlock, "AddTakeGradLargeBatch");
  nnvm::dim_t* lookup_table = prefix_sum;

  switch (SZ) {
    case 1:
    AddTakeGradLargeBatchKernel<1, true, DType>
        <<<dimGrid, dimBlock, shmem_size, stream>>>
        (dst.dptr_, sum_counts_ptr, num_runs_ptr,
         sorted.dptr_, index.dptr_, src.dptr_,
         static_cast<int>(src.size(0)),
         static_cast<int>(src.size(1)), lookup_table);
    break;
    case 2:
    AddTakeGradLargeBatchKernel<2, true, DType>
        <<<dimGrid, dimBlock, shmem_size, stream>>>
        (dst.dptr_, sum_counts_ptr, num_runs_ptr,
         sorted.dptr_, index.dptr_, src.dptr_,
         static_cast<int>(src.size(0)),
         static_cast<int>(src.size(1)), lookup_table);
    break;
    case 3:
    AddTakeGradLargeBatchKernel<3, true, DType>
        <<<dimGrid, dimBlock, shmem_size, stream>>>
        (dst.dptr_, sum_counts_ptr, num_runs_ptr,
         sorted.dptr_, index.dptr_, src.dptr_,
         static_cast<int>(src.size(0)),
         static_cast<int>(src.size(1)), lookup_table);
    break;
    case 4:
    AddTakeGradLargeBatchKernel<4, true, DType>
        <<<dimGrid, dimBlock, shmem_size, stream>>>
        (dst.dptr_, sum_counts_ptr, num_runs_ptr,
         sorted.dptr_, index.dptr_, src.dptr_,
         static_cast<int>(src.size(0)),
         static_cast<int>(src.size(1)), lookup_table);
    break;
    default:
    LOG(FATAL) << "AddTakeGradLargeBatch, incorrect value SZ " << SZ;
    break;
  }
  MSHADOW_CUDA_POST_KERNEL_CHECK(AddTakeGradLargeBatchKernel);


}


   s->Wait();
  auto t5 = std::chrono::duration_cast<std::chrono::microseconds>(
    std::chrono::high_resolution_clock::now().time_since_epoch()).count();
bool log = dmlc::GetEnv("LOG_TIME", true);
if (log) {
LOG(INFO) << t1-t0 << "\t"
          << t2-t1 << "\t"
          << t3-t2 << "\t"
          << t4-t3 << "\t"
          << t5-t4 << "\t"
          << t5-t0 << "\t";
}
      });
    });
  });
}

template<>
inline void SparseEmbeddingOpBackwardRspImpl<gpu>(const OpContext& ctx,
                                                  const TBlob& ograd,
                                                  const TBlob& data,
                                                  const OpReqType req,
                                                  const NDArray& output) {
  
  int deterministic = dmlc::GetEnv("MXNET_DETERM", 0);
  if (deterministic == 1) {
    LOG(FATAL) << "NOT REAHCED";
    return;
  } else if (deterministic == 2) {
    SparseEmbeddingOpBackwardRspImpl3(ctx, ograd, data, req, output);
    return;
  }
  using namespace mshadow;
  using namespace mxnet_op;
  using namespace mshadow::expr;
  using namespace rowsparse;
  using nnvm::dim_t;
  if (req == kNullOp) return;
  CHECK_EQ(req, kWriteTo) << "SparseEmbedding layer doesn't support "
                          << "weight gradient calculation with req != write";

  // Request temporary storage for marking non-zero rows and prefix sum
  Stream<gpu> *s = ctx.get_stream<gpu>();
  dim_t num_rows = output.shape()[0];
  dim_t row_length = output.shape()[1];
  dim_t data_size = static_cast<dim_t>(data.shape_.Size());
  dim_t num_threads;

  MSHADOW_TYPE_SWITCH(data.type_flag_, IType, {
    MSHADOW_SGL_DBL_TYPE_SWITCH(ograd.type_flag_, DType, {
      MSHADOW_IDX_TYPE_SWITCH(output.aux_type(kIdx), RType, {
        dim_t* prefix_sum = NULL;
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      Stream<gpu>::GetStream(s));
        Tensor<gpu, 1, char> workspace = ctx.requested[0]
            .get_space_typed<gpu, 1, char>(Shape1(num_rows * sizeof(dim_t) +
                                           temp_storage_bytes), s);
        prefix_sum = reinterpret_cast<dim_t*>(workspace.dptr_);
        d_temp_storage = workspace.dptr_ + num_rows*sizeof(dim_t);
        num_threads = num_rows;
        Fill<false>(s, TBlob(prefix_sum, Shape1(num_threads), gpu::kDevMask), kWriteTo, 0);
        Kernel<MarkRowFlgKernel, gpu>::Launch(s, data_size, prefix_sum, data.dptr<IType>());

        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      mshadow::Stream<gpu>::GetStream(s));
        dim_t nnr = 0;
        CUDA_CALL(hipMemcpy(&nnr, &prefix_sum[num_rows-1], sizeof(dim_t),
            hipMemcpyDeviceToHost));

        if (nnr == 0) {
          FillZerosRspImpl(s, output);
          return;
        }
        output.CheckAndAlloc({Shape1(nnr)});
        RType* grad_row_idx = output.aux_data(kIdx).dptr<RType>();
        // fill row_idx array of output matrix, using the row_flg values
        Kernel<FillRspRowIdxKernel, gpu>::Launch(s, num_rows,
            grad_row_idx, prefix_sum, num_rows);
        // prefill with zeros
        DType* grad_data = output.data().dptr<DType>();
        Fill<false>(s, TBlob(grad_data, Shape1(nnr * row_length), gpu::kDevMask),
            kWriteTo, 0);
        // add the final gradients
        num_threads = row_length * data_size;
        Kernel<AddTakeGradRspGPUKernel, gpu>::Launch(s, num_threads, grad_data, prefix_sum,
            data.dptr<IType>(), ograd.dptr<DType>(), row_length);
      });
    });
  });
}

struct backward_gather_nd_gpu {
  template<typename DType, typename IType>
  MSHADOW_XINLINE static void Map(int i, int N, int M, int K,
                                  const mshadow::Shape<10> strides,
                                  DType* out, const DType* data,
                                  const IType* indices) {
    int offset = 0;
    for (int j = 0; j < M; ++j) {
      offset += strides[j] * static_cast<int>(indices[j*N + i]);
    }
    for (int j = 0; j < K; ++j) {
      atomicAdd(out + (offset + j), data[i * K + j]);
    }
  }
};

template<typename DType, typename IType>
inline void GatherNDBackwardImpl(int N, int M, int K,
                                 const mshadow::Shape<10> strides,
                                 DType* out,
                                 const DType* data,
                                 const IType* indices,
                                 mshadow::Stream<gpu> *s) {
  mxnet_op::Kernel<backward_gather_nd_gpu, gpu>::Launch(s, N, N, M, K, strides, out, data, indices);
}

NNVM_REGISTER_OP(Embedding)
.set_attr<FCompute>("FCompute<gpu>", EmbeddingOpForward<gpu>);

NNVM_REGISTER_OP(_contrib_SparseEmbedding)
.set_attr<FComputeEx>("FComputeEx<gpu>", SparseEmbeddingOpForwardEx<gpu>);

NNVM_REGISTER_OP(_backward_Embedding)
.set_attr<FCompute>("FCompute<gpu>", EmbeddingOpBackward<gpu>);

NNVM_REGISTER_OP(_backward_SparseEmbedding)
.set_attr<FComputeEx>("FComputeEx<gpu>", SparseEmbeddingOpBackwardEx<gpu>);

NNVM_REGISTER_OP(take)
.set_attr<FCompute>("FCompute<gpu>", TakeOpForward<gpu>);

NNVM_REGISTER_OP(_backward_take)
.set_attr<FCompute>("FCompute<gpu>", TakeOpBackward<gpu>);

NNVM_REGISTER_OP(batch_take)
.set_attr<FCompute>("FCompute<gpu>", BatchTakeOpForward<gpu>);

NNVM_REGISTER_OP(one_hot)
.set_attr<FCompute>("FCompute<gpu>", OneHotOpForward<gpu>);

NNVM_REGISTER_OP(gather_nd)
.set_attr<FCompute>("FCompute<gpu>", GatherNDForward<gpu>);

NNVM_REGISTER_OP(scatter_nd)
.set_attr<FCompute>("FCompute<gpu>", ScatterNDForward<gpu>);

NNVM_REGISTER_OP(_backward_gather_nd)
.set_attr<FCompute>("FCompute<gpu>", GatherNDBackward<gpu>);

NNVM_REGISTER_OP(_scatter_set_nd)
.set_attr<FCompute>("FCompute<gpu>", ScatterSetNDForward<gpu>);
}  // namespace op
}  // namespace mxnet
