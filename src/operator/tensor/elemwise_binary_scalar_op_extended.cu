#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2016 by Contributors
 * \file elemwise_binary_scalar_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_unary_op.h"
#include "./elemwise_binary_op.h"
#include "./elemwise_binary_scalar_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(_backward_rdiv_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::rdiv_grad>);

NNVM_REGISTER_OP(_power_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::power>);

NNVM_REGISTER_OP(_backward_power_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::power_grad>);

NNVM_REGISTER_OP(_rpower_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::rpower>);

NNVM_REGISTER_OP(_backward_rpower_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::rpower_grad>);

NNVM_REGISTER_OP(_hypot_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::hypot>);

NNVM_REGISTER_OP(_backward_hypot_scalar)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::hypot_grad_left>);

NNVM_REGISTER_OP(smooth_l1)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarCompute<gpu, mshadow_op::smooth_l1_loss>);

NNVM_REGISTER_OP(_backward_smooth_l1)
.set_attr<FCompute>("FCompute<gpu>", BinaryScalarBackward<gpu, mshadow_op::smooth_l1_gradient>);

}  // namespace op
}  // namespace mxnet

