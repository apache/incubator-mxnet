#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2020 by Contributors
 * \file quantized_bert_ffn1_to_ffn2.cu
 * \brief
*/
#include "../quantization_utils.h"
#include "../../mxnet_op.h"
#include "../../nn/fully_connected-inl.h"
#include "./quantized_bert_ffn1_to_ffn2-inl.h"

namespace mxnet {
namespace op {

#if CUDA_VERSION >= 8000

// get float_for_one_out_quant and float_for_one_bias_quant outside QuantizedBiasAddKernel
struct FloatForOneQuantBiasAddKernel {
  MSHADOW_XINLINE static void Map(int i,
                                  const float *min_out,
                                  const float *max_out, 
                                  const float *min_bias,
                                  const float *max_bias,
                                  float *float_for_one_quant_tmp) {
    typedef int32_t T1;
    typedef int8_t  T2;
    
    float float_for_one_quant_out = FloatForOneQuantizedLevel<T1>(*min_out, *max_out, true);
    float float_for_one_quant_bias = FloatForOneQuantizedLevel<T2>(*min_bias, *max_bias, true);

    // the tmp space to store float_for_one_quant is 32 bits (1 float numbers)
    *float_for_one_quant_tmp = float_for_one_quant_bias / float_for_one_quant_out;
  }
};

// get quantizedtofloat scale
struct QuantizedToFloatScale {
  MSHADOW_XINLINE static void Map(int i,
                                  const float *min_out,
                                  const float *max_out, 
                                  float *quantized_to_float_scale) {
    typedef int32_t T1;

    float quantized_range = MinAbs(MinValue<T1>(), MaxValue<T1>());
    float real_range = MaxAbs(*min_out, *max_out);
    float scale = real_range / quantized_range;

    // the tmp space to store quantized_to_float_scale is 32 bits (1 float numbers)
    *quantized_to_float_scale = scale;
  }
};

// get quantization_from_float_scale and set omin_range, omax_range for quantize_v2_zero_centered
struct QuantizedFromFloatScale {
  MSHADOW_XINLINE static void Map(int i,
                                float *omin_range,
                                float *omax_range,
                                const float imin_range,
                                const float imax_range,
                                float *quantization_from_float_scale){

    float real_range = MaxAbs(imin_range, imax_range);
    float scale = MinAbs(MaxValue<int8_t>(), MinValue<int8_t>()) / real_range;

    *omin_range = -real_range;
    *omax_range = real_range;

    // the tmp space to store quantization_from_float_scale is 32 bits (1 float numbers)
    *quantization_from_float_scale = scale;
  }
};
#endif  // CUDA_VERSION >= 8000

#if defined(__HIPCC__)

const float SQRT_2 = 1.4142135623730950488016887242096;
// compatible with mshadow_op.h version
template <typename DType>
__device__ inline DType gelu(const DType val) {
  return DType(0.5f * static_cast<float>(val) *
               (1.0f + erf(static_cast<float>(val) / SQRT_2)));
}

// value + bias_value * (range1 / limit_range1) * (limit_range2 / range2)
// DType->output type (int8_t), BType->bias type (int8_t)
__global__ void quantized_add_bias_redequantize_gelu_quantize_kernel(int32_t* outCUBLAS,
                                          int8_t* out,
                                          int8_t* bias,
                                          size_t bias_length,
                                          const float *float_for_one_quant_tmp,
                                          const float *quantized_to_float_scale,
                                          const float *quantization_from_float_scale,
                                          const float quantized_range) {
  int64_t* outCUBLASload = reinterpret_cast<int64_t*>(outCUBLAS);
  int16_t* outload = reinterpret_cast<int16_t*>(out);
  int16_t* biasload = reinterpret_cast<int16_t*>(bias);
  
  int row_num_each_thread = 8;
  
  for (index_t i = threadIdx.x; i < bias_length / 2; i += blockDim.x){

    int16_t scratch_bias = *(biasload + i);
    int8_t* scratch_bias_aft_load = reinterpret_cast<int8_t*>(&scratch_bias);

  #pragma unroll
    for(int rw = 0; rw < row_num_each_thread; rw++){
      int idx = (blockIdx.x * row_num_each_thread + rw) * bias_length / 2 + i;

      int64_t scratch_outCUBLAS = *(outCUBLASload + idx);
      int32_t* scratch_outCUBLAS_aft_load = reinterpret_cast<int32_t*>(&scratch_outCUBLAS);

      int16_t scratch_out = *(outload + idx);
      int8_t* scratch_out_aft_load = reinterpret_cast<int8_t*>(&scratch_out);

      // add_bias and dequantize to float
      mshadow::half::half_t FCout = (scratch_outCUBLAS_aft_load[0] + scratch_bias_aft_load[0] * (*float_for_one_quant_tmp)) * 
                                      (*quantized_to_float_scale);
      // gelu
      mshadow::half::half_t geluout = gelu<mshadow::half::half_t>(FCout);
      // quantize it back to int8
      scratch_out_aft_load[0] = Sign(geluout) * fminf(fabsf(geluout) * (*quantization_from_float_scale) + 0.5f, quantized_range);


      // add_bias and dequantize to float
      FCout = (scratch_outCUBLAS_aft_load[1] + scratch_bias_aft_load[1] * (*float_for_one_quant_tmp)) * 
                                      (*quantized_to_float_scale);
      // gelu
      geluout = gelu<mshadow::half::half_t>(FCout);
      // quantize it back to int8
      scratch_out_aft_load[1] = Sign(geluout) * fminf(fabsf(geluout) * (*quantization_from_float_scale) + 0.5f, quantized_range);


      *(outload + idx) = scratch_out;
    }
  }
}

void FusedQuantizedAddBias_ReDequantize_GELU_Quantize(int8_t* bias,
                      Tensor<gpu, 2, int8_t> data,
                      int8_t* out,
                      Tensor<gpu, 2, int32_t> outTensorCUBLAS,
                      Stream<gpu>* s,
                      const float *float_for_one_quant_tmp,
                      const int bias_len,
                      const float *quantized_to_float_scale,
                      const float *quantization_from_float_scale) {
    
    int nthreads_quant_addbias = 256;

    if(bias_len % 512 == 0){
      nthreads_quant_addbias = 512;
    }else if(bias_len % 256 == 0){
      nthreads_quant_addbias = 256;
    }else if(bias_len % 128 == 0){
      nthreads_quant_addbias = 128;
    }else if(bias_len % 64 == 0){
      nthreads_quant_addbias = 64;
    }
    
    if(bias_len <= 32){
      nthreads_quant_addbias = 32;
    }

    float quantized_range = MinAbs(MaxValue<int8_t>(), MinValue<int8_t>());

    quantized_add_bias_redequantize_gelu_quantize_kernel<<<data.size(0) / 8, //row_num_each_thread = 8
                                  nthreads_quant_addbias,
                                  0,
                                  Stream<gpu>::GetStream(s)>>>(outTensorCUBLAS.dptr_,
                                                                out,
                                                                bias,
                                                                bias_len,
                                                                float_for_one_quant_tmp,
                                                                quantized_to_float_scale,
                                                                quantization_from_float_scale,
                                                                quantized_range);
}

#endif  // __HIPCC__

template<typename SrcType>
void QuantizedBERTFFN1TOFFN2ForwardGPU(const nnvm::NodeAttrs& attrs,
                                       const OpContext &ctx,
                                       const std::vector<TBlob> &inputs,
                                       const std::vector<OpReqType> &req,
                                       const std::vector<TBlob> &outputs) {
#if CUDA_VERSION >= 8000
  typedef int32_t CmpType;

  const QuantizedBERTFFN1TOFFN2Param& param = nnvm::get<QuantizedBERTFFN1TOFFN2Param>(attrs.parsed);
  using namespace mshadow;
  using namespace mxnet_op;
  size_t num_inputs = 3;
  CHECK_EQ(inputs.size(),  num_inputs * 3);
  CHECK_EQ(outputs.size(), 3U);
  Stream<gpu> *s = ctx.get_stream<gpu>();
  CHECK_EQ(s->blas_handle_ownership_, Stream<gpu>::OwnHandle);

  const TBlob& data   =  inputs[0];
  const TBlob& weight =  inputs[1];
  const TBlob& out    = outputs[0];

  mxnet::TShape dshape = data.shape_;
  mxnet::TShape wshape = weight.shape_;
  mxnet::TShape oshape = out.shape_;
  // (m, n) * (k, n).T = (m, k)
  // A * B.T = C

  // allocate workspace for storaging both outTensorCUBLAS and FloatForOneQuant, QuantizedToFloatScaleFactor and 
  //  QuantizedFromFloatScaleFactor
  size_t workspace_size = sizeof(int32_t) * out.Size() + sizeof(float) * 3;
  auto workspace = ctx.requested[0].get_space_typed<gpu, 1, char>(Shape1(workspace_size), s);
  char* ptr = workspace.dptr_;

  Tensor<gpu, 2, SrcType> dataTensor;

  Tensor<gpu, 2, int8_t> outTensor;
    if (!param.flatten) {
      dataTensor = FlattenAs2DHead<gpu, SrcType>(data, ctx);
      outTensor = FlattenAs2DHead<gpu, int8_t>(out, ctx);
    } else {
      dataTensor = FlattenAs2DTail<gpu, SrcType>(data, ctx);
      outTensor = FlattenAs2DTail<gpu, int8_t>(out, ctx);
    }

    // workspace: temporary storage for output tensor, which is in int32 type for storaging CUBLAS output
    Tensor<gpu, 2, int32_t> outTensorCUBLAS = Tensor<gpu, 2, int32_t>(reinterpret_cast<int32_t*>(ptr), outTensor.shape_, s);
    ptr += sizeof(int32_t) * out.Size();
    // workspace: FloatForOneQuant
    Tensor<gpu, 1, float> FloatForOneQuant = Tensor<gpu, 1, float>(reinterpret_cast<float*>(ptr), Shape1(1), s);
    ptr += sizeof(float);
    // workspace: QuantizedToFloatScaleFactor
    Tensor<gpu, 1, float> QuantizedToFloatScaleFactor = Tensor<gpu, 1, float>(reinterpret_cast<float*>(ptr), Shape1(1), s);
    ptr += sizeof(float);
    // workspace: QuantizedFromFloatScaleFactor
    Tensor<gpu, 1, float> QuantizedFromFloatScaleFactor = Tensor<gpu, 1, float>(reinterpret_cast<float*>(ptr), Shape1(1), s);

  Tensor<gpu, 2, SrcType> weightTensor = weight.get<gpu, 2, SrcType>(s);

  // row_C = col_C(T) = cublas(col_B * col_A(T)) = cublas(row_B(T), row_A)
  // row_C = col_C(T) = cublas(col_B(T) * col_A(T)) = cublas(row_B, row_A)
 
  // A->dataTensor, B->weightTensor, C->outTensor
  CmpType alpha = 1.0f;
  CmpType beta  = 0.0f;
  const hipDataType src_type = mshadow::DataType<SrcType>::kCudaFlag;
  const hipDataType dst_type = mshadow::DataType<int32_t>::kCudaFlag;
  const hipDataType cmp_type = mshadow::DataType<CmpType>::kCudaFlag;
  CUBLAS_CALL(hipblasGemmEx(s->blas_handle_,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_N,
                           outTensorCUBLAS.size(1),
                           outTensorCUBLAS.size(0),
                           weightTensor.size(1),
                           &alpha,
                           weightTensor.dptr_,
                           src_type,
                           weightTensor.stride_,
                           dataTensor.dptr_,
                           src_type,
                           dataTensor.stride_,
                           &beta,
                           outTensorCUBLAS.dptr_,
                           dst_type,
                           outTensorCUBLAS.stride_,
                           cmp_type,
                           HIPBLAS_GEMM_DEFAULT));
  
  // use min/max values of weight and data to update the min/max values of output
  Kernel<QuantizationRangeForS8S8MultiplicationStruct, gpu>::Launch(s, 1,
    outputs[1].dptr<float>(), outputs[2].dptr<float>(),
     inputs[num_inputs].dptr<float>(),   inputs[num_inputs+1].dptr<float>(),
     inputs[num_inputs+2].dptr<float>(), inputs[num_inputs+3].dptr<float>());

  // Launch QuantizedToFloatScale
  Kernel<QuantizedToFloatScale, gpu>::Launch(s, 1,
      outputs[1].dptr<float>(), outputs[2].dptr<float>(),
      QuantizedToFloatScaleFactor.dptr_);

  const TBlob& bias = inputs[2];

  Tensor<gpu, 1, SrcType> biasTensor = bias.get_with_shape<gpu, 1, SrcType>(Shape1(wshape[0]), s);
  CHECK_EQ(biasTensor.shape_[0], wshape[0])
      << "Incomplete bias tensor detected: bias.data().shape[1] != weight.data().shape[0]."
         " This is not supported by FCForward. If bias is in row_sparse format, please"
         " make sure all row ids are present.";

  // Launch FloatForOneQuantBiasAddKernel
  Kernel<FloatForOneQuantBiasAddKernel, gpu>::Launch(s, 1,
      outputs[1].dptr<float>(), outputs[2].dptr<float>(),
      inputs[7].dptr<float>(), inputs[8].dptr<float>(),
      FloatForOneQuant.dptr_);

  // // to calcualte the quantization_from_float_scale and reset the outputs[1] and outputs[2]
  Kernel<QuantizedFromFloatScale, gpu>::Launch(s, 1,
      outputs[1].dptr<float>(), outputs[2].dptr<float>(),
      param.min_calib_range.value(), param.max_calib_range.value(),
      QuantizedFromFloatScaleFactor.dptr_);

  // with_bias case with float16 out
  // a kernel that fuse requantize, dequantize into add_bias
  FusedQuantizedAddBias_ReDequantize_GELU_Quantize(bias.dptr<SrcType>(), dataTensor,
                                          out.dptr<int8_t>(), outTensorCUBLAS, s,
                                          FloatForOneQuant.dptr_, biasTensor.shape_[0],
                                          QuantizedToFloatScaleFactor.dptr_,
                                          QuantizedFromFloatScaleFactor.dptr_);

#else
  LOG(FATAL) << "QuantizedBERTFFN1TOFFN2ForwardGPU only supports CUDA >= 8.0";
#endif  // CUDA_VERSION >= 8000
}

NNVM_REGISTER_OP(_contrib_quantized_bert_ffn1_to_ffn2_fusion)
.set_attr<FCompute>("FCompute<gpu>", QuantizedBERTFFN1TOFFN2ForwardGPU<int8_t>);

}  // namespace op
}  // namespace mxnet