#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2017 by Contributors
 * \file utils.cu
 * \brief gpu implementation of util functions
 */
#include "./utils.h"
#include "../common/utils.h"
#include <hipcub/hipcub.hpp>
#include <mxnet/resource.h>

namespace mxnet {
namespace kvstore {

/*!
 * \brief sort and get unique values.
 */
template<>
void UniqueImpl<gpu>(const Resource& rsc, mshadow::Stream<gpu> *s,
                     NDArray *out, nnvm::dim_t size) {
  MSHADOW_IDX_TYPE_SWITCH(out->data().type_flag_, IType, {
    IType *dptr = out->data().dptr<IType>();
    size_t sort_temp_bytes = 0;
    hipcub::DeviceRadixSort::SortKeys(NULL, sort_temp_bytes,
      dptr, dptr, size, 0, sizeof(IType)*8, mshadow::Stream<gpu>::GetStream(s));
    size_t unique_temp_bytes = 0;
    mshadow::Tensor<gpu, 1, char> dummy_space = rsc
      .get_space_typed<gpu, 1, char>(
        mshadow::Shape1(sizeof(size_t)), s);
    size_t *dummy_ptr = reinterpret_cast<size_t*>(dummy_space.dptr_);
    hipcub::DeviceSelect::Unique(NULL, unique_temp_bytes, dptr, dptr,
      dummy_ptr, size, mshadow::Stream<gpu>::GetStream(s));

    size_t cub_temp_bytes = std::max(sort_temp_bytes, unique_temp_bytes);
    mshadow::Tensor<gpu, 1, char> workspace = rsc
      .get_space_typed<gpu, 1, char>(
        mshadow::Shape1((cub_temp_bytes + sizeof(size_t) + 7) / 8 * 8), s);

    void *sort_temp_storage = static_cast<void*>(workspace.dptr_);
    void *unique_temp_storage = static_cast<void*>(
      workspace.dptr_);
    size_t *d_num_selected_out = reinterpret_cast<size_t*>(
      workspace.dptr_ + (cub_temp_bytes + 7) / 8 * 8);

    hipcub::DeviceRadixSort::SortKeys(sort_temp_storage, sort_temp_bytes,
      dptr, dptr, size, 0, sizeof(IType)*8, mshadow::Stream<gpu>::GetStream(s));

    hipcub::DeviceSelect::Unique(unique_temp_storage, unique_temp_bytes, dptr, dptr,
      d_num_selected_out, size, mshadow::Stream<gpu>::GetStream(s));
    s->Wait();

    size_t num_selected_out = 0;
    CUDA_CALL(hipMemcpy(&num_selected_out, d_num_selected_out, sizeof(size_t),
       hipMemcpyDeviceToHost));
    *out = out->Reshape(mshadow::Shape1(num_selected_out));
  });
}


}  // namespace kvstore
}  // namespace mxnet
