#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2017 by Contributors
 *  \file cp_decomp_test.cc
 *  \brief Test CPDecomp core function
 *  \author Jencir Lee
 */
#include <mxnet/tensor_blob.h>
#include <gtest/gtest.h>
#include <vector>
#include "operator/tensor/unfold.h"

namespace mxnet {
namespace op {

using namespace std;
using namespace mshadow;
using DType = double;

__global__
void AccessElements(int n, const DType *dptr, DType *result, const int *indices) {
  for (int i = 0; i < n; ++i)
    result[i] = dptr[indices[i]];
}


TEST(Unfold, ravel_multi_index_2D_gpu) {
  Tensor<gpu, 2, DType> ts(Shape2(19, 29));
  Tensor<cpu, 2, DType> ts_cpu(ts.shape_);
  AllocSpace(&ts);
  AllocSpace(&ts_cpu);

  int *indices;
  DType *result;
  hipMallocManaged(reinterpret_cast<void **>(&indices),
      ts.shape_.Size() * sizeof(int));
  hipMallocManaged(reinterpret_cast<void **>(&result),
      ts.shape_.Size() * sizeof(DType));

  Shape<2> strides = ts.shape_;
  strides[1] = ts.stride_;

  int c = 0;
  Shape<2> coord;
  for (int i = 0; i < static_cast<int>(ts.size(0)); ++i)
    for (int j = 0; j < static_cast<int>(ts.size(1)); ++j) {
      coord[0] = i;
      coord[1] = j;
      indices[c] = ravel_multi_index(coord, strides);

      ts_cpu[i][j] = ++c;
    }
  hipDeviceSynchronize();

  hipMemcpy2D(ts.dptr_, ts.stride_ * sizeof(DType),
      ts_cpu.dptr_, ts_cpu.stride_ * sizeof(DType),
      ts.size(1) * sizeof(DType), ts.size(0),
      hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  AccessElements<<<1, 1>>>(ts.shape_.Size(), ts.dptr_, result, indices);
  hipDeviceSynchronize();

  for (int i = 0; i < static_cast<int>(ts.shape_.Size()); ++i) {
    EXPECT_DOUBLE_EQ(result[i], i + 1);
  }

  FreeSpace(&ts);
  FreeSpace(&ts_cpu);
  hipFree(result);
  hipFree(indices);
}

TEST(Unfold, ravel_multi_index_3D_gpu) {
  Tensor<gpu, 3, DType> ts(Shape3(19, 29, 53));
  Tensor<cpu, 3, DType> ts_cpu(ts.shape_);
  AllocSpace(&ts);
  AllocSpace(&ts_cpu);

  int *indices;
  DType *result;
  hipMallocManaged(reinterpret_cast<void **>(&indices), ts.shape_.Size() * sizeof(int));
  hipMallocManaged(reinterpret_cast<void **>(&result), ts.shape_.Size() * sizeof(DType));

  Shape<3> strides = ts.shape_;
  strides[2] = ts.stride_;

  int c = 0;
  Shape<3> coord;
  for (int i = 0; i < static_cast<int>(ts.size(0)); ++i)
    for (int j = 0; j < static_cast<int>(ts.size(1)); ++j)
      for (int k = 0; k < static_cast<int>(ts.size(2)); ++k) {
        coord[0] = i;
        coord[1] = j;
        coord[2] = k;
        indices[c] = ravel_multi_index(coord, strides);

        ts_cpu[i][j][k] = ++c;
      }
  hipDeviceSynchronize();

  hipMemcpy2D(ts.dptr_, ts.stride_ * sizeof(DType),
      ts_cpu.dptr_, ts_cpu.stride_ * sizeof(DType),
      ts.size(2) * sizeof(DType), ts.size(0) * ts.size(1),
      hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  AccessElements<<<1, 1>>>(ts.shape_.Size(), ts.dptr_, result, indices);
  hipDeviceSynchronize();

  for (int i = 0; i < static_cast<int>(ts.shape_.Size()); ++i) {
    EXPECT_DOUBLE_EQ(result[i], i + 1);
  }

  FreeSpace(&ts);
  FreeSpace(&ts_cpu);
  hipFree(result);
  hipFree(indices);
}

}  // namespace op
}  // namespace mxnet
